#include "Particles.h"
#include "ParticlesBatching.h"
#include "ParticlesStreaming.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define NUMBER_OF_PARTICLES_PER_BATCH 1024000
#define MAX_NUMBER_OF_STREAMS 5
#define NUMBER_OF_STREAMS_PER_BATCH 4


/** particle mover for GPU with batching */

int mover_GPU_stream(struct particles* part, struct EMfield* field, struct grid* grd, struct parameters* param)
{
    // print species and subcycling
    std::cout << "***GPU MOVER with SUBCYCLYING "<< param->n_sub_cycles << " - species " << part->species_ID << " ***" << std::endl;

    // auxiliary variables
    FPpart dt_sub_cycling = (FPpart) param->dt/((double) part->n_sub_cycles);
    FPpart dto2 = .5*dt_sub_cycling, qomdt2 = part->qom*dto2/param->c;

    // allocate memory for variables on device

    FPpart *x_dev = NULL, *y_dev = NULL, *z_dev = NULL, *u_dev = NULL, *v_dev = NULL, *w_dev = NULL;
    FPinterp *q_dev = NULL;
    FPfield *XN_flat_dev = NULL, *YN_flat_dev = NULL, *ZN_flat_dev = NULL, *Ex_flat_dev = NULL, *Ey_flat_dev = NULL, *Ez_flat_dev = NULL, *Bxn_flat_dev = NULL, *Byn_flat_dev, *Bzn_flat_dev = NULL;

    size_t free_bytes = 0;

    int i, total_size_particles, start_index_batch, end_index_batch, number_of_batches;

    // Calculation done later to compute free space after allocating space on the GPU fo
    // other variables below, the assumption is that these variables fit in the GPU memory 
    // and mini batching is implemented only taking into account particles

    hipMalloc(&XN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&YN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&ZN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Ex_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Ey_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Ez_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Bxn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Byn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Bzn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));

    hipMemcpy(XN_flat_dev, grd->XN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(YN_flat_dev, grd->YN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(ZN_flat_dev, grd->ZN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Ex_flat_dev, field->Ex_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Ey_flat_dev, field->Ey_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Ez_flat_dev, field->Ez_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Bxn_flat_dev, field->Bxn_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Byn_flat_dev, field->Byn_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Bzn_flat_dev, field->Bzn_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    free_bytes = queryFreeMemoryOnGPU();
    total_size_particles = sizeof(FPpart) * part->npmax * 6 + sizeof(FPinterp) * part->npmax; //  for x,y,z,u,v,w and q
    
    start_index_batch = 0, end_index_batch = 0;

    // implement mini-batching only in the case where the free space on the GPU isn't enough

    if(free_bytes > total_size_particles)
    {
        start_index_batch = 0;
        end_index_batch = part->npmax - 1; // set end_index to the last particle as we are processing in in one batch
        number_of_batches = 1;
    }
    else
    {
        start_index_batch = 0;
        end_index_batch = start_index_batch + NUMBER_OF_PARTICLES_PER_BATCH - 1; // NUM_PARTICLES_PER_BATCH is a hyperparameter set by tuning
        if(part->npmax % NUMBER_OF_PARTICLES_PER_BATCH != 0)
        {
            number_of_batches = part->npmax / NUMBER_OF_PARTICLES_PER_BATCH + 1; // works because of integer division
        }
        else
        {
            number_of_batches = part->npmax / NUMBER_OF_PARTICLES_PER_BATCH;
        }
    }


    hipStream_t cudaStreams[MAX_NUMBER_OF_STREAMS];

    for(i = 0; i < number_of_batches; i++)
    {

        long int number_of_particles_batch = end_index_batch - start_index_batch + 1; // number of particles in  a batch
        size_t batch_size_per_attribute = number_of_particles_batch * sizeof(FPpart); // size of the attribute per batch in bytes x,z,y,u,v,w

        long int number_of_particles_stream = 0, stream_size_per_attribute = 0, number_of_streams = 0, stream_offset = 0, offset = 0, start_index_stream = 0, end_index_stream = 0, max_num_particles_per_stream = 0;

        int flag_leftover = 0;

        hipMalloc(&x_dev, batch_size_per_attribute);
        hipMalloc(&y_dev, batch_size_per_attribute);
        hipMalloc(&z_dev, batch_size_per_attribute);
        hipMalloc(&u_dev, batch_size_per_attribute);
        hipMalloc(&v_dev, batch_size_per_attribute);
        hipMalloc(&w_dev, batch_size_per_attribute);
        hipMalloc(&q_dev, number_of_particles_batch * sizeof(FPinterp));

        start_index_stream = 0;
        end_index_stream = start_index_stream + (number_of_particles_batch / NUMBER_OF_STREAMS_PER_BATCH) - 1;
        max_num_particles_per_stream = number_of_particles_batch / NUMBER_OF_STREAMS_PER_BATCH;            

        if(number_of_particles_batch % NUMBER_OF_STREAMS_PER_BATCH != 0) // We have some leftover bytes
        {
            number_of_streams = NUMBER_OF_STREAMS_PER_BATCH;
        }
        else
        {
            number_of_streams = NUMBER_OF_STREAMS_PER_BATCH;
        }

        for (int j = 0; j < number_of_streams; j++)
        {
            hipStreamCreate(&cudaStreams[j]);
        }

        for (int stream_idx = 0; stream_idx < number_of_streams; stream_idx++)
        {

            number_of_particles_stream = end_index_stream - start_index_stream + 1;
            stream_size_per_attribute = number_of_particles_stream * sizeof(FPpart); // for x,y,z,u,v,w

            stream_offset = start_index_stream;
            offset = stream_offset + start_index_batch; // batch offset + stream_offset

            hipMemcpyAsync(&x_dev[stream_offset], &part->x[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&y_dev[stream_offset], &part->y[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&z_dev[stream_offset], &part->z[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&u_dev[stream_offset], &part->u[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&v_dev[stream_offset], &part->v[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&w_dev[stream_offset], &part->w[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&q_dev[stream_offset], &part->q[offset], number_of_particles_stream * sizeof(FPinterp), hipMemcpyHostToDevice, cudaStreams[stream_idx]);

            // start subcycling
            for (int i_sub=0; i_sub < part->n_sub_cycles; i_sub++){

            // Call GPU kernel

                single_particle_kernel<<<(number_of_particles_stream + TPB - 1)/TPB, TPB, 0, cudaStreams[stream_idx]>>>(
                    x_dev, y_dev, z_dev, u_dev, v_dev, w_dev, q_dev, 
                    XN_flat_dev, YN_flat_dev, ZN_flat_dev, 
                    grd->nxn, grd->nyn, grd->nzn, 
                    grd->xStart, grd->yStart, grd->zStart, 
                    grd->invdx, grd->invdy, grd->invdz, 
                    grd->Lx, grd->Ly, grd->Lz, grd->invVOL, 
                    Ex_flat_dev, Ey_flat_dev, Ez_flat_dev, 
                    Bxn_flat_dev, Byn_flat_dev, Bzn_flat_dev, 
                    param->PERIODICX, param->PERIODICY, param->PERIODICZ, 
                    dt_sub_cycling, dto2, qomdt2, 
                    part->NiterMover, number_of_particles_stream, stream_offset
                );

            } // end of one particle


            hipMemcpyAsync(&part->x[offset], &x_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->y[offset], &y_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->z[offset], &z_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->u[offset], &u_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->v[offset], &v_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->w[offset], &w_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->q[offset], &q_dev[stream_offset], number_of_particles_stream * sizeof(FPinterp), hipMemcpyDeviceToHost, cudaStreams[stream_idx]);

            hipStreamSynchronize(cudaStreams[stream_idx]);

            start_index_stream = start_index_stream + max_num_particles_per_stream;
    
            if( (start_index_stream + max_num_particles_per_stream) > number_of_particles_batch)
            {
                end_index_stream = number_of_particles_batch - 1;
            }
            else
            {
                end_index_stream += max_num_particles_per_stream;
            } 

        }

        for(int j = 0; j < number_of_streams; j++)
        {
            hipStreamDestroy(cudaStreams[j]);
        }

        hipFree(x_dev);
        hipFree(y_dev);
        hipFree(z_dev);
        hipFree(u_dev);
        hipFree(v_dev);
        hipFree(w_dev);
        hipFree(q_dev);

        // Update indices for next batch
        start_index_batch = start_index_batch + NUMBER_OF_PARTICLES_PER_BATCH;
    
        if( (start_index_batch + NUMBER_OF_PARTICLES_PER_BATCH) > part->npmax)
        {
            end_index_batch = part->npmax - 1;
        }
        else
        {
            end_index_batch += NUMBER_OF_PARTICLES_PER_BATCH;
        }

    }
        
    hipMemcpy(field->Ex_flat, Ex_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Ey_flat, Ey_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Ez_flat, Ez_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Bxn_flat, Bxn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Byn_flat, Byn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Bzn_flat, Bzn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    
    // Clean up
    hipFree(XN_flat_dev);
    hipFree(YN_flat_dev);
    hipFree(ZN_flat_dev);
    hipFree(Ex_flat_dev);
    hipFree(Ey_flat_dev);
    hipFree(Ez_flat_dev);
    hipFree(Bxn_flat_dev);
    hipFree(Byn_flat_dev);
    hipFree(Bzn_flat_dev);

    return(0);
}

/** Interpolation with batching */

void interpP2G_GPU_stream(struct particles* part, struct interpDensSpecies* ids, struct grid* grd)
{

    FPpart *x_dev = NULL, *y_dev = NULL, *z_dev = NULL, *u_dev = NULL, *v_dev = NULL, *w_dev = NULL;
    FPinterp * q_dev = NULL, *Jx_flat_dev = NULL, *Jy_flat_dev = NULL, *Jz_flat_dev = NULL, *rhon_flat_dev = NULL, *pxx_flat_dev = NULL, *pxy_flat_dev = NULL, *pxz_flat_dev = NULL, *pyy_flat_dev = NULL, *pyz_flat_dev = NULL, *pzz_flat_dev = NULL;
    FPfield *XN_flat_dev = NULL, *YN_flat_dev = NULL, *ZN_flat_dev = NULL;

    size_t free_bytes = 0;

    int i, total_size_particles, start_index_batch, end_index_batch, number_of_batches;

    // Calculation done later to compute free space after allocating space on the GPU for 
    // other variables below, the assumption is that these variables fit in the GPU memory 
    // and mini batching is implemented only taking into account particles

    hipMalloc(&Jx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&Jy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&Jz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&rhon_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pxx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pxy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pxz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pyy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pyz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pzz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&XN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&YN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&ZN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));

    hipMemcpy(Jx_flat_dev, ids->Jx_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Jy_flat_dev, ids->Jy_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Jz_flat_dev, ids->Jz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(rhon_flat_dev, ids->rhon_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pxx_flat_dev, ids->pxx_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pxy_flat_dev, ids->pxy_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pxz_flat_dev, ids->pxz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pyy_flat_dev, ids->pyy_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pyz_flat_dev, ids->pyz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pzz_flat_dev, ids->pzz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(XN_flat_dev, grd->XN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(YN_flat_dev, grd->YN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(ZN_flat_dev, grd->ZN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    free_bytes = queryFreeMemoryOnGPU();
    total_size_particles = sizeof(FPpart) * part->npmax * 6 + sizeof(FPinterp) * part->npmax; // for x,y,z,u,v,w and q
    
    start_index_batch = 0, end_index_batch = 0;

    // implement mini-batching only in the case where the free space on the GPU isn't enough

    if(free_bytes > total_size_particles)
    {
        start_index_batch = 0;
        end_index_batch = part->npmax - 1 ; // set end_index to the last particle as we are processing in in one batch
        number_of_batches = 1;
    }
    else
    {
        start_index_batch = 0;
        end_index_batch = start_index_batch + NUMBER_OF_PARTICLES_PER_BATCH - 1; // NUM_PARTICLES_PER_BATCH is a hyperparameter set by tuning
        number_of_batches = part->npmax / NUMBER_OF_PARTICLES_PER_BATCH + 1; // works because of integer division
    }

    hipStream_t cudaStreams[MAX_NUMBER_OF_STREAMS];

    for(i = 0; i < number_of_batches; i++)
    {

        long int number_of_particles_batch = end_index_batch - start_index_batch + 1; // number of particles in  a batch
        size_t batch_size = number_of_particles_batch * sizeof(FPpart); // size of the batch in bytes

        long int number_of_particles_stream = 0, stream_size_per_attribute = 0, number_of_streams = 0, stream_offset = 0, offset = 0, start_index_stream = 0, end_index_stream = 0, max_num_particles_per_stream = 0;

        hipMalloc(&x_dev, batch_size);
        hipMalloc(&y_dev, batch_size);
        hipMalloc(&z_dev, batch_size);
        hipMalloc(&u_dev, batch_size);
        hipMalloc(&v_dev, batch_size);
        hipMalloc(&w_dev, batch_size);
        hipMalloc(&q_dev, number_of_particles_batch * sizeof(FPinterp));

        start_index_stream = 0;
        end_index_stream = start_index_stream + (number_of_particles_batch / NUMBER_OF_STREAMS_PER_BATCH) - 1;
        max_num_particles_per_stream = number_of_particles_batch / NUMBER_OF_STREAMS_PER_BATCH;            

        if(number_of_particles_batch % NUMBER_OF_STREAMS_PER_BATCH != 0) // We have some leftover bytes
        {
            number_of_streams = NUMBER_OF_STREAMS_PER_BATCH + 1;
            flag_leftover = 1;
        }
        else
        {
            number_of_streams = NUMBER_OF_STREAMS_PER_BATCH;
        }

        for (int j = 0; j < number_of_streams; j++)
        {
            hipStreamCreate(&cudaStreams[j]);
        }

        for (int stream_idx = 0; stream_idx < number_of_streams; stream_idx++)
        {

            number_of_particles_stream = end_index_stream - start_index_stream + 1;
            stream_size_per_attribute = number_of_particles_stream * sizeof(FPpart); // for x,y,z,u,v,w

            stream_offset = start_index_stream;
            offset = stream_offset + start_index_batch; // batch offset + stream_offset

            hipMemcpyAsync(&x_dev[stream_offset], &part->x[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&y_dev[stream_offset], &part->y[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&z_dev[stream_offset], &part->z[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&u_dev[stream_offset], &part->u[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&v_dev[stream_offset], &part->v[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&w_dev[stream_offset], &part->w[offset], stream_size_per_attribute, hipMemcpyHostToDevice, cudaStreams[stream_idx]);
            hipMemcpyAsync(&q_dev[stream_offset], &part->q[offset], number_of_particles_stream * sizeof(FPinterp), hipMemcpyHostToDevice, cudaStreams[stream_idx]);

            // Call GPU kernel
            interP2G_kernel<<<(number_of_particles_stream + TPB - 1)/TPB, TPB, 0, cudaStreams[stream_idx]>>>(
                x_dev, y_dev, z_dev, u_dev, v_dev, w_dev, q_dev, 
                XN_flat_dev, YN_flat_dev, ZN_flat_dev, 
                grd->nxn, grd->nyn, grd->nzn, 
                grd->xStart, grd->yStart, grd->zStart, 
                grd->invdx, grd->invdy, grd->invdz, grd->invVOL, 
                Jx_flat_dev, Jy_flat_dev, Jz_flat_dev, rhon_flat_dev, 
                pxx_flat_dev , pxy_flat_dev, pxz_flat_dev, pyy_flat_dev, pyz_flat_dev, pzz_flat_dev, 
                number_of_particles_stream, stream_offset
            );

            hipMemcpyAsync(&part->x[offset], &x_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->y[offset], &y_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->z[offset], &z_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->u[offset], &u_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->v[offset], &v_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);
            hipMemcpyAsync(&part->w[offset], &w_dev[stream_offset], stream_size_per_attribute, hipMemcpyDeviceToHost, cudaStreams[stream_idx]);                

            hipStreamSynchronize(cudaStreams[stream_idx]);

            start_index_stream = start_index_stream + max_num_particles_per_stream;
    
            if( (start_index_stream + max_num_particles_per_stream) > number_of_particles_batch)
            {
                end_index_stream = number_of_particles_batch - 1;
            }
            else
            {
                end_index_stream += max_num_particles_per_stream;
            } 

        }

        for(int j = 0; j < number_of_streams; j++)
        {
            hipStreamDestroy(cudaStreams[j]);
        }

        hipFree(x_dev);
        hipFree(y_dev);
        hipFree(z_dev);
        hipFree(u_dev);
        hipFree(v_dev);
        hipFree(w_dev);
        hipFree(q_dev);

        // Update indices for next batch
        start_index_batch = start_index_batch + NUMBER_OF_PARTICLES_PER_BATCH;
        if ((start_index_batch + NUMBER_OF_PARTICLES_PER_BATCH) > part->npmax)
        {
            end_index_batch = part->npmax - 1;
        }
        else
        {
            end_index_batch += NUMBER_OF_PARTICLES_PER_BATCH;
        }

    }

    // Copy memory back to CPU (only the parts that have been modified inside the kernel)
    hipMemcpy(ids->Jx_flat, Jx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->Jy_flat, Jy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->Jz_flat, Jz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->rhon_flat, rhon_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxx_flat, pxx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxy_flat, pxy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxz_flat, pxz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pyy_flat, pyy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pyz_flat, pyz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pzz_flat, pzz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    
    // Clean up
    hipFree(Jx_flat_dev);
    hipFree(Jy_flat_dev);
    hipFree(Jz_flat_dev);
    hipFree(XN_flat_dev);
    hipFree(YN_flat_dev);
    hipFree(ZN_flat_dev);
    hipFree(rhon_flat_dev);
    hipFree(pxx_flat_dev);
    hipFree(pxy_flat_dev);
    hipFree(pxz_flat_dev);
    hipFree(pyy_flat_dev);
    hipFree(pyz_flat_dev);
    hipFree(pzz_flat_dev);

}